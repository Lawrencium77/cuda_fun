#include "hip/hip_runtime.h"
#include <stdio.h>
#include <pybind11/pybind11.h>
#include "add.h"

float add_wrapper(int N)
{
  float *x, *y, *out, *d_x, *d_y, *d_out;
  x = (float *)malloc(N * sizeof(float));
  y = (float *)malloc(N * sizeof(float));
  out = (float *)malloc(N * sizeof(float));

  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));
  hipMalloc(&d_out, N * sizeof(float));

  for (int i = 0; i < N; i++)
  {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  // Execute the kernel
  add<<<1, 1>>>(N, d_x, d_y, d_out);

  hipMemcpy(out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, fabs(out[i] - 3.0));

  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_out);
  free(x);
  free(y);
  free(out);

  return maxError;
}

PYBIND11_MODULE(example, m)
{
  m.def("add_static_arrays", &add_wrapper, "A function which adds two arrays");
}