
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void add(int n, float *x, float *y, float *out)
{
  for (int i = 0; i < n; i++) {
    out[i] = x[i] + y[i];
  }
}